#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <Windows.h>
#include <ctime>
#define BLOCKS				2048
//#define BLOCKS              8192
//#define BLOCKS				64
#define THREADS				1024

__int64 trial = 1, keys = 10;   double PCFreq = 0.0;    __int64 CounterStart = 0;
#define bit8 unsigned char
#define bit16 unsigned short int
#define bit32 unsigned int
#define bit64 unsigned __int64 
bit8 S7[128] = {   54, 50, 62, 56, 22, 34, 94, 96, 38,  6, 63, 93,  2, 18,123, 33,   55,113, 39,114, 21, 67, 65, 12, 47, 73, 46, 27, 25,111,124, 81,   53,  9,121, 79, 52, 60, 58, 48,101,127, 40,120,104, 70, 71, 43,   20,122, 72, 61, 23,109, 13,100, 77,  1, 16,  7, 82, 10,105, 98,  117,116, 76, 11, 89,106,  0,125,118, 99, 86, 69, 30, 57,126, 87,  112, 51, 17,  5, 95, 14, 90, 84, 91,  8, 35,103, 32, 97, 28, 66,  102, 31, 26, 45, 75,  4, 85, 92, 37, 74, 80, 49, 68, 29,115, 44,   64,107,108, 24,110, 83, 36, 78, 42, 19, 15, 41, 88,119, 59,  3};
bit16 S9[512] = {  167,239,161,379,391,334,  9,338, 38,226, 48,358,452,385, 90,397,  183,253,147,331,415,340, 51,362,306,500,262, 82,216,159,356,177,  175,241,489, 37,206, 17,  0,333, 44,254,378, 58,143,220, 81,400,   95,  3,315,245, 54,235,218,405,472,264,172,494,371,290,399, 76,  165,197,395,121,257,480,423,212,240, 28,462,176,406,507,288,223,  501,407,249,265, 89,186,221,428,164, 74,440,196,458,421,350,163,  232,158,134,354, 13,250,491,142,191, 69,193,425,152,227,366,135,  344,300,276,242,437,320,113,278, 11,243, 87,317, 36, 93,496, 27,  487,446,482, 41, 68,156,457,131,326,403,339, 20, 39,115,442,124,  475,384,508, 53,112,170,479,151,126,169, 73,268,279,321,168,364,  363,292, 46,499,393,327,324, 24,456,267,157,460,488,426,309,229,  439,506,208,271,349,401,434,236, 16,209,359, 52, 56,120,199,277,  465,416,252,287,246,  6, 83,305,420,345,153,502, 65, 61,244,282,  173,222,418, 67,386,368,261,101,476,291,195,430, 49, 79,166,330,  280,383,373,128,382,408,155,495,367,388,274,107,459,417, 62,454,  132,225,203,316,234, 14,301, 91,503,286,424,211,347,307,140,374,   35,103,125,427, 19,214,453,146,498,314,444,230,256,329,198,285,   50,116, 78,410, 10,205,510,171,231, 45,139,467, 29, 86,505, 32,   72, 26,342,150,313,490,431,238,411,325,149,473, 40,119,174,355, 185,233,389, 71,448,273,372, 55,110,178,322, 12,469,392,369,190,   1,109,375,137,181, 88, 75,308,260,484, 98,272,370,275,412,111, 336,318,  4,504,492,259,304, 77,337,435, 21,357,303,332,483, 18,  47, 85, 25,497,474,289,100,269,296,478,270,106, 31,104,433, 84,  414,486,394, 96, 99,154,511,148,413,361,409,255,162,215,302,201,  266,351,343,144,441,365,108,298,251, 34,182,509,138,210,335,133,  311,352,328,141,396,346,123,319,450,281,429,228,443,481, 92,404,  485,422,248,297, 23,213,130,466, 22,217,283, 70,294,360,419,127,  312,377,  7,468,194,  2,117,295,463,258,224,447,247,187, 80,398,  284,353,105,390,299,471,470,184, 57,200,348, 63,204,188, 33,451,   97, 30,310,219, 94,160,129,493, 64,179,263,102,189,207,114,402,  438,477,387,122,192, 42,381,  5,145,118,180,449,293,323,136,380,   43, 66, 60,455,341,445,202,432,  8,237, 15,376,436,464, 59,461};
__device__ bit32 arithmeticRightShift(bit32 x, bit32 n) { return (x >> n) | (x << (-n & 31)); }
bit16 LeftShift(bit16 x, bit16 n) { return (x << n) | (x >> (-n & 15)); }
__device__ bit16 LeftShiftd(bit16 x, bit16 n) { return (x << n) | (x >> (-n & 15)); }
__shared__ bit8 S7S[128];
__shared__ bit16 S9S[512];
__shared__ bit16 constants[8];
bit16 constant[8] ={ 0x0123, 0x4567, 0x89AB,0xCDEF, 0xFEDC, 0xBA98, 0x7654, 0x3210};
//__shared__ bit16 S9S2[256][32][2];

bit16 FI(bit16 input, bit16 roundkey) {
    bit16 left, right, round_key_1, round_key_2, tmp_l, tmp_r;
    left = input >> 7;
    right = input & 0b1111111;
    round_key_1 = roundkey >> 9;
    round_key_2 = roundkey & 0b111111111;
    tmp_l = right;
    tmp_r = S9[left] ^ right;
    left = tmp_r ^ round_key_2;
    right = S7[tmp_l] ^ (tmp_r & 0b1111111) ^ round_key_1;
    tmp_l = right;
    tmp_r = S9[left] ^ right;
    left = S7[tmp_l] ^ (tmp_r & 0b1111111);
    right = tmp_r;
    return (left << 9) | right;
}
__device__ bit16 FId(bit16 input, bit16 roundkey) {
    bit16 left, right, round_key_1, round_key_2, tmp_l, tmp_r;
    left = input >> 7;
    right = input & 0b1111111;
    round_key_1 = roundkey >> 9;
    round_key_2 = roundkey & 0b111111111;
    tmp_l = right;
    tmp_r = S9S[left] ^ right;
    left = tmp_r ^ round_key_2;
    right = S7S[tmp_l] ^ (tmp_r & 0b1111111) ^ round_key_1;
    tmp_l = right;
    tmp_r = S9S[left] ^ right;
    left = S7S[tmp_l] ^ (tmp_r & 0b1111111);
/*    right = tmp_r;
    return (left << 9) | right;*/
    return (left << 9) | tmp_r;
}
/*__device__ bit16 FId(bit16 input, bit16 roundkey) {
    bit16 left, right, round_key_1, round_key_2, tmp_l, tmp_r;
    int warpThreadIndex = threadIdx.x & 31;
    left = input >> 7;
    right = input & 0b1111111;
    round_key_1 = roundkey >> 9;
    round_key_2 = roundkey & 0b111111111;
    tmp_l = right;
    tmp_r = S9S2[left/2][warpThreadIndex][left%2] ^ right;
    left = tmp_r ^ round_key_2;
    right = S7S[tmp_l] ^ (tmp_r & 0b1111111) ^ round_key_1;
    tmp_l = right;
    tmp_r = S9S2[left/2][warpThreadIndex][left%2] ^ right;
    left = S7S[tmp_l] ^ (tmp_r & 0b1111111);
    right = tmp_r;
    return (left << 9) | right;
}*/
bit32 FO(bit32 input, bit16 KO1, bit16 KO2, bit16 KO3, bit16 KI1, bit16 KI2, bit16 KI3) {
    bit16 in_left, in_right, out_left, out_right;
    in_left = input >> 16;
    in_right = input & 0xFFFF;
    out_left = in_right;
    out_right = FI(in_left ^ KO1, KI1) ^ in_right;
    in_left = out_right;
    in_right = FI(out_left ^ KO2, KI2) ^ out_right;
    out_left = in_right;
    out_right = FI(in_left ^ KO3, KI3) ^ in_right;
    return (out_left << 16) | out_right;
}
__device__ bit32 FOd(bit32 input, bit16 KO1, bit16 KO2, bit16 KO3, bit16 KI1, bit16 KI2, bit16 KI3) {
    bit16 in_left, in_right, out_left, out_right;
    in_left = input >> 16;
    in_right = input & 0xFFFF;
    out_left = in_right;
    out_right = FId(in_left ^ KO1, KI1) ^ in_right;
    in_left = out_right;
    in_right = FId(out_left ^ KO2, KI2) ^ out_right;
    out_left = in_right;
    out_right = FId(in_left ^ KO3, KI3) ^ in_right;
    return (out_left << 16) | out_right;
}
bit32 FL(bit32 input, bit16 KL1, bit16 KL2) {
    bit32 in_left, in_right, out_right, out_left;
    in_left = input >> 16;
    in_right = input & 0xFFFF;
    out_right = in_right ^ LeftShift(in_left & KL1, 1);
    out_left = in_left ^ LeftShift(out_right | KL2, 1);
    return (out_left << 16) | out_right;
}
__device__ bit32 FLd(bit32 input, bit16 KL1, bit16 KL2) {
    bit16 in_left, in_right, out_right, out_left;
    in_left = input >> 16;
    in_right = input & 0xFFFF;
    out_right = in_right ^ LeftShiftd(in_left & KL1, 1);
    out_left = in_left ^ LeftShiftd(out_right | KL2, 1);
    return (out_left << 16) | out_right;
}
void encryption(bit32 left, bit32 right, bit32 cipher_left, bit32 cipher_right) {
    bit32 in_left = left, in_right = right, temp;
    bit16 k1= 0, k2=0, k3=0, k4=0, k5=0, k6=0, k7=0, k8=0;
    bit16 KL1, KL2, KO1, KO2, KO3, KI1, KI2, KI3;
    //    C1 0x0123
    //    C2 0x4567
    //    C3 0x89AB
    //    C4 0xCDEF
    //    C5 0xFEDC
    //    C6 0xBA98
    //    C7 0x7654
    //    C8 0x3210
    // Round 1
    KL1 = LeftShift(k1, 1);    KL2 = k3 ^ 0x89AB;
    KO1 = LeftShift(k2, 5);    KO2 = LeftShift(k6, 8);    KO3 = LeftShift(k7, 13);
    KI1 = k5 ^ 0xFEDC;    KI2 = k4 ^ 0xCDEF;    KI3 = k8 ^ 0x3210;
    temp = FL(in_left, KL1, KL2);
    temp = FO(temp, KO1, KO2, KO3, KI1, KI2, KI3);
    temp ^= in_right;    in_right = in_left;    in_left = temp;
    // Round 2
    KL1 = LeftShift(k2, 1);    KL2 = k4 ^ 0xCDEF;
    KO1 = LeftShift(k3, 5);    KO2 = LeftShift(k7, 8);    KO3 = LeftShift(k8, 13);
    KI1 = k6 ^ 0xBA98;    KI2 = k5 ^ 0xFEDC;    KI3 = k1 ^ 0x0123;
    temp = FO(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
    temp = FL(temp, KL1, KL2);    
    temp ^= in_right;    in_right = in_left;    in_left = temp;
    // Round 3
    KL1 = LeftShift(k3, 1);    KL2 = k5 ^ 0xFEDC;
    KO1 = LeftShift(k4, 5);    KO2 = LeftShift(k8, 8);    KO3 = LeftShift(k1, 13);
    KI1 = k7 ^ 0x7654;    KI2 = k6 ^ 0xBA98;    KI3 = k2 ^ 0x4567;
    temp = FL(in_left, KL1, KL2);
    temp = FO(temp, KO1, KO2, KO3, KI1, KI2, KI3);
    temp ^= in_right;    in_right = in_left;    in_left = temp;
    // Round 4
    KL1 = LeftShift(k4, 1);    KL2 = k6 ^ 0xBA98;
    KO1 = LeftShift(k5, 5);    KO2 = LeftShift(k1, 8);    KO3 = LeftShift(k2, 13);
    KI1 = k8 ^ 0x3210;    KI2 = k7 ^ 0x7654;    KI3 = k3 ^ 0x89AB;
    temp = FO(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
    temp = FL(temp, KL1, KL2);
    temp ^= in_right;    in_right = in_left;    in_left = temp;
    // Round 5
    KL1 = LeftShift(k5, 1);    KL2 = k7 ^ 0x7654;
    KO1 = LeftShift(k6, 5);    KO2 = LeftShift(k2, 8);    KO3 = LeftShift(k3, 13);
    KI1 = k1 ^ 0x0123;    KI2 = k8 ^ 0x3210;    KI3 = k4 ^ 0xCDEF;
    temp = FL(in_left, KL1, KL2);
    temp = FO(temp, KO1, KO2, KO3, KI1, KI2, KI3);
    temp ^= in_right;    in_right = in_left;    in_left = temp;
    // Round 6
    KL1 = LeftShift(k6, 1);    KL2 = k8 ^ 0x3210;
    KO1 = LeftShift(k7, 5);    KO2 = LeftShift(k3, 8);    KO3 = LeftShift(k4, 13);
    KI1 = k2 ^ 0x4567;    KI2 = k1 ^ 0x0123;    KI3 = k5 ^ 0xFEDC;
    temp = FO(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
    temp = FL(temp, KL1, KL2);
    temp ^= in_right;    in_right = in_left;    in_left = temp;
    // Round 7
    KL1 = LeftShift(k7, 1);    KL2 = k1 ^ 0x0123;
    KO1 = LeftShift(k8, 5);    KO2 = LeftShift(k4, 8);    KO3 = LeftShift(k5, 13);
    KI1 = k3 ^ 0x89AB;    KI2 = k2 ^ 0x4567;    KI3 = k6 ^ 0xBA98;
    temp = FL(in_left, KL1, KL2);
    temp = FO(temp, KO1, KO2, KO3, KI1, KI2, KI3);
    temp ^= in_right;    in_right = in_left;    in_left = temp;
    // Round 8
    KL1 = LeftShift(k8, 1);    KL2 = k2 ^ 0x4567;
    KO1 = LeftShift(k1, 5);    KO2 = LeftShift(k5, 8);    KO3 = LeftShift(k6, 13);
    KI1 = k4 ^ 0xCDEF;    KI2 = k3 ^ 0x89AB;    KI3 = k7 ^ 0x7654;
    temp = FO(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
    temp = FL(temp, KL1, KL2);
    temp ^= in_right;    in_right = in_left;    in_left = temp;
    printf("%08x%08x\n", in_left, in_right);


    if (in_left == cipher_left)
        if (in_right == cipher_right)
            printf("Can you see me?\n"); 
}
__global__ void KASUMI64Exhaustive(bit32 left, bit32 right, bit32 cipher_left, bit32 cipher_right, bit8* S7G, bit16* S9G) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
//    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 512) {
        if (threadIdx.x < 128) S7S[threadIdx.x] = S7G[threadIdx.x];
        S9S[threadIdx.x] = S9G[threadIdx.x];
    }
    __syncthreads();
    bit32 in_left, in_right, temp;
    bit16 k1 = threadIndex/65536, k2 = threadIndex % 65536, k3 = 0, k4 = 0, k5 = k1, k6 = k2, k7 = 0, k8 = 0;
    bit16 KL1, KL2, KO1, KO2, KO3, KI1, KI2, KI3;
//#pragma unroll
    for (int j = 0; j < 1; j++) {
        for (int i = 0; i < 65536; i++) {
            in_left = left; in_right = right;
            // Round 1
            KL1 = LeftShiftd(k1, 1);    KL2 = k3 ^ 0x89AB;
            KO1 = LeftShiftd(k2, 5);    KO2 = LeftShiftd(k6, 8);    KO3 = LeftShiftd(k7, 13);
            KI1 = k5 ^ 0xFEDC;    KI2 = k4 ^ 0xCDEF;    KI3 = k8 ^ 0x3210;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 2
            KL1 = LeftShiftd(k2, 1);    KL2 = k4 ^ 0xCDEF;
            KO1 = LeftShiftd(k3, 5);    KO2 = LeftShiftd(k7, 8);    KO3 = LeftShiftd(k8, 13);
            KI1 = k6 ^ 0xBA98;    KI2 = k5 ^ 0xFEDC;    KI3 = k1 ^ 0x0123;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 3
            KL1 = LeftShiftd(k3, 1);    KL2 = k5 ^ 0xFEDC;
            KO1 = LeftShiftd(k4, 5);    KO2 = LeftShiftd(k8, 8);    KO3 = LeftShiftd(k1, 13);
            KI1 = k7 ^ 0x7654;    KI2 = k6 ^ 0xBA98;    KI3 = k2 ^ 0x4567;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 4
            KL1 = LeftShiftd(k4, 1);    KL2 = k6 ^ 0xBA98;
            KO1 = LeftShiftd(k5, 5);    KO2 = LeftShiftd(k1, 8);    KO3 = LeftShiftd(k2, 13);
            KI1 = k8 ^ 0x3210;    KI2 = k7 ^ 0x7654;    KI3 = k3 ^ 0x89AB;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 5
            KL1 = LeftShiftd(k5, 1);    KL2 = k7 ^ 0x7654;
            KO1 = LeftShiftd(k6, 5);    KO2 = LeftShiftd(k2, 8);    KO3 = LeftShiftd(k3, 13);
            KI1 = k1 ^ 0x0123;    KI2 = k8 ^ 0x3210;    KI3 = k4 ^ 0xCDEF;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 6
            KL1 = LeftShiftd(k6, 1);    KL2 = k8 ^ 0x3210;
            KO1 = LeftShiftd(k7, 5);    KO2 = LeftShiftd(k3, 8);    KO3 = LeftShiftd(k4, 13);
            KI1 = k2 ^ 0x4567;    KI2 = k1 ^ 0x0123;    KI3 = k5 ^ 0xFEDC;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 7
            KL1 = LeftShiftd(k7, 1);    KL2 = k1 ^ 0x0123;
            KO1 = LeftShiftd(k8, 5);    KO2 = LeftShiftd(k4, 8);    KO3 = LeftShiftd(k5, 13);
            KI1 = k3 ^ 0x89AB;    KI2 = k2 ^ 0x4567;    KI3 = k6 ^ 0xBA98;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;
            if (temp == cipher_right) {
                in_right = in_left;    in_left = temp;
                // Round 8
                KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ 0x4567;
                KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
                KI1 = k4 ^ 0xCDEF;    KI2 = k3 ^ 0x89AB;    KI3 = k7 ^ 0x7654;
                temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
                temp = FLd(temp, KL1, KL2);
                temp ^= in_right; //   in_right = in_left;    in_left = temp;
                if (temp == cipher_left) printf("The secret key is %08x%08x\n", threadIndex, i);
            }
            k8++; k4 = k8;
        }
        k7++; k3 = k7;
        /*       KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ 0x4567;
        KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
        KI1 = k4 ^ 0xCDEF;    KI2 = k3 ^ 0x89AB;    KI3 = k7 ^ 0x7654;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right; //   in_right = in_left;    in_left = temp;
        //printf("%08x%08x\n", in_left, in_right);

        if (temp == cipher_left)
            if (in_left == cipher_right)
                printf("The secret key is %08x%08x\n", threadIndex, i);
        k8++; k4 = k8;*/
    }
}
__global__ void KASUMI64EncryptionTMTO(bit32 left, bit32 right, bit32 cipher_left, bit32 cipher_right, bit8* S7G, bit16* S9G) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 512) {
        if (threadIdx.x < 128) S7S[threadIdx.x] = S7G[threadIdx.x];
        S9S[threadIdx.x] = S9G[threadIdx.x];
    }
    __syncthreads();
    bit32 in_left, in_right, temp;
    bit16 k1 = threadIndex / 65536, k2 = threadIndex % 65536, k3 = 0, k4 = 0, k5 = k1, k6 = k2, k7 = 0, k8 = 0;
    bit16 KL1, KL2, KO1, KO2, KO3, KI1, KI2, KI3;
    //#pragma unroll
    for (int j = 0; j < 1; j++) {
        for (int i = 0; i < 65536; i++) {
            in_left = left; in_right = right;
            // Round 1
            KL1 = LeftShiftd(k1, 1);    KL2 = k3 ^ 0x89AB;
            KO1 = LeftShiftd(k2, 5);    KO2 = LeftShiftd(k6, 8);    KO3 = LeftShiftd(k7, 13);
            KI1 = k5 ^ 0xFEDC;    KI2 = k4 ^ 0xCDEF;    KI3 = k8 ^ 0x3210;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 2
            KL1 = LeftShiftd(k2, 1);    KL2 = k4 ^ 0xCDEF;
            KO1 = LeftShiftd(k3, 5);    KO2 = LeftShiftd(k7, 8);    KO3 = LeftShiftd(k8, 13);
            KI1 = k6 ^ 0xBA98;    KI2 = k5 ^ 0xFEDC;    KI3 = k1 ^ 0x0123;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 3
            KL1 = LeftShiftd(k3, 1);    KL2 = k5 ^ 0xFEDC;
            KO1 = LeftShiftd(k4, 5);    KO2 = LeftShiftd(k8, 8);    KO3 = LeftShiftd(k1, 13);
            KI1 = k7 ^ 0x7654;    KI2 = k6 ^ 0xBA98;    KI3 = k2 ^ 0x4567;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 4
            KL1 = LeftShiftd(k4, 1);    KL2 = k6 ^ 0xBA98;
            KO1 = LeftShiftd(k5, 5);    KO2 = LeftShiftd(k1, 8);    KO3 = LeftShiftd(k2, 13);
            KI1 = k8 ^ 0x3210;    KI2 = k7 ^ 0x7654;    KI3 = k3 ^ 0x89AB;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 5
            KL1 = LeftShiftd(k5, 1);    KL2 = k7 ^ 0x7654;
            KO1 = LeftShiftd(k6, 5);    KO2 = LeftShiftd(k2, 8);    KO3 = LeftShiftd(k3, 13);
            KI1 = k1 ^ 0x0123;    KI2 = k8 ^ 0x3210;    KI3 = k4 ^ 0xCDEF;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 6
            KL1 = LeftShiftd(k6, 1);    KL2 = k8 ^ 0x3210;
            KO1 = LeftShiftd(k7, 5);    KO2 = LeftShiftd(k3, 8);    KO3 = LeftShiftd(k4, 13);
            KI1 = k2 ^ 0x4567;    KI2 = k1 ^ 0x0123;    KI3 = k5 ^ 0xFEDC;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 7
            KL1 = LeftShiftd(k7, 1);    KL2 = k1 ^ 0x0123;
            KO1 = LeftShiftd(k8, 5);    KO2 = LeftShiftd(k4, 8);    KO3 = LeftShiftd(k5, 13);
            KI1 = k3 ^ 0x89AB;    KI2 = k2 ^ 0x4567;    KI3 = k6 ^ 0xBA98;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;
 
                in_right = in_left;    in_left = temp;
                // Round 8
                KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ 0x4567;
                KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
                KI1 = k4 ^ 0xCDEF;    KI2 = k3 ^ 0x89AB;    KI3 = k7 ^ 0x7654;
                temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
                temp = FLd(temp, KL1, KL2);
                temp ^= in_right; //   in_right = in_left;    in_left = temp;
                if (temp == cipher_left && in_left ==cipher_right) printf("The secret key is %08x%08x\n", threadIndex, i);
            
            k8++; k4 = k8;
        }
        k7++; k3 = k7;
        /*       KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ 0x4567;
        KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
        KI1 = k4 ^ 0xCDEF;    KI2 = k3 ^ 0x89AB;    KI3 = k7 ^ 0x7654;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right; //   in_right = in_left;    in_left = temp;
        //printf("%08x%08x\n", in_left, in_right);

        if (temp == cipher_left)
            if (in_left == cipher_right)
                printf("The secret key is %08x%08x\n", threadIndex, i);
        k8++; k4 = k8;*/
    }
}
__global__ void KASUMI64ExhaustiveConstantsRegister(bit32 left, bit32 right, bit32 cipher_left, bit32 cipher_right, bit8* S7G, bit16* S9G) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 512) {
        if (threadIdx.x < 128) S7S[threadIdx.x] = S7G[threadIdx.x];
        S9S[threadIdx.x] = S9G[threadIdx.x];
    }
    __syncthreads();
    bit32 in_left, in_right, temp;
    bit16 k1 = threadIndex / 65536, k2 = threadIndex % 65536, k3 = 0, k4 = 0, k5 = k1, k6 = k2, k7 = 0, k8 = 0;
    bit16 KL1, KL2, KO1, KO2, KO3, KI1, KI2, KI3;
    bit16 c1 = 0x0123, c2 = 0x4567, c3 = 0x89AB, c4 = 0xCDEF, c5 = 0xFEDC, c6 = 0xBA98, c7 = 0x7654, c8 = 0x3210;

#pragma unroll
    for (int j = 0; j < 16; j++) {
    for (int i = 0; i < 65536; i++) {
        in_left = left; in_right = right;
        // Round 1
        KL1 = LeftShiftd(k1, 1);    KL2 = k3 ^ c3;
        KO1 = LeftShiftd(k2, 5);    KO2 = LeftShiftd(k6, 8);    KO3 = LeftShiftd(k7, 13);
        KI1 = k5 ^ c5;    KI2 = k4 ^ c4;    KI3 = k8 ^ c8;
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 2
        KL1 = LeftShiftd(k2, 1);    KL2 = k4 ^ c4;
        KO1 = LeftShiftd(k3, 5);    KO2 = LeftShiftd(k7, 8);    KO3 = LeftShiftd(k8, 13);
        KI1 = k6 ^ c6;    KI2 = k5 ^ c5;    KI3 = k1 ^ c1;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 3
        KL1 = LeftShiftd(k3, 1);    KL2 = k5 ^ c5;
        KO1 = LeftShiftd(k4, 5);    KO2 = LeftShiftd(k8, 8);    KO3 = LeftShiftd(k1, 13);
        KI1 = k7 ^ c7;    KI2 = k6 ^ c6;    KI3 = k2 ^ c2;
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 4
        KL1 = LeftShiftd(k4, 1);    KL2 = k6 ^ c6;
        KO1 = LeftShiftd(k5, 5);    KO2 = LeftShiftd(k1, 8);    KO3 = LeftShiftd(k2, 13);
        KI1 = k8 ^ c8;    KI2 = k7 ^ c7;    KI3 = k3 ^ c3;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 5
        KL1 = LeftShiftd(k5, 1);    KL2 = k7 ^ c7;
        KO1 = LeftShiftd(k6, 5);    KO2 = LeftShiftd(k2, 8);    KO3 = LeftShiftd(k3, 13);
        KI1 = k1 ^ c1;    KI2 = k8 ^ c8;    KI3 = k4 ^ c4;
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 6
        KL1 = LeftShiftd(k6, 1);    KL2 = k8 ^ c8;
        KO1 = LeftShiftd(k7, 5);    KO2 = LeftShiftd(k3, 8);    KO3 = LeftShiftd(k4, 13);
        KI1 = k2 ^ c2;    KI2 = k1 ^ c1;    KI3 = k5 ^ c5;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 7
        KL1 = LeftShiftd(k7, 1);    KL2 = k1 ^ c1;
        KO1 = LeftShiftd(k8, 5);    KO2 = LeftShiftd(k4, 8);    KO3 = LeftShiftd(k5, 13);
        KI1 = k3 ^ c3;    KI2 = k2 ^ c2;    KI3 = k6 ^ c6;
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;
        if (temp == cipher_right) {
            in_right = in_left;    in_left = temp;
            // Round 8
            KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ c2;
            KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
            KI1 = k4 ^ c4;    KI2 = k3 ^ c3;    KI3 = k7 ^ c7;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right; //   in_right = in_left;    in_left = temp;
            if (temp == cipher_left) printf("The secret key is %08x%08x\n", threadIndex, i);
        }
        k8++; k4 = k8;
    }
    k7++; k3 = k7;
        /*       KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ 0x4567;
        KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
        KI1 = k4 ^ 0xCDEF;    KI2 = k3 ^ 0x89AB;    KI3 = k7 ^ 0x7654;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right; //   in_right = in_left;    in_left = temp;
        //printf("%08x%08x\n", in_left, in_right);

        if (temp == cipher_left)
            if (in_left == cipher_right)
                printf("The secret key is %08x%08x\n", threadIndex, i);
        k8++; k4 = k8;*/
    }
}
__global__ void KASUMI64ExhaustiveConstantsRegisterTMTO(bit32 left, bit32 right, bit32 cipher_left, bit32 cipher_right, bit8* S7G, bit16* S9G, bit32 cipherl[], bit32 cipherr[]) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 512) {
        if (threadIdx.x < 128) S7S[threadIdx.x] = S7G[threadIdx.x];
        S9S[threadIdx.x] = S9G[threadIdx.x];
    }
    __syncthreads();
    bit32 in_left, in_right, temp;
    bit16 k1 = threadIndex / 65536, k2 = threadIndex % 65536, k3 = 0, k4 = 0, k5 = k1, k6 = k2, k7 = 0, k8 = 0;
    bit16 KL1, KL2, KO1, KO2, KO3, KI1, KI2, KI3;
    bit16 c1 = 0x0123, c2 = 0x4567, c3 = 0x89AB, c4 = 0xCDEF, c5 = 0xFEDC, c6 = 0xBA98, c7 = 0x7654, c8 = 0x3210;

#pragma unroll
    for (int j = 0; j < 16; j++) {
        for (int i = 0; i < 65536; i++) {
            in_left = left; in_right = right;
            // Round 1
            KL1 = LeftShiftd(k1, 1);    KL2 = k3 ^ c3;
            KO1 = LeftShiftd(k2, 5);    KO2 = LeftShiftd(k6, 8);    KO3 = LeftShiftd(k7, 13);
            KI1 = k5 ^ c5;    KI2 = k4 ^ c4;    KI3 = k8 ^ c8;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 2
            KL1 = LeftShiftd(k2, 1);    KL2 = k4 ^ c4;
            KO1 = LeftShiftd(k3, 5);    KO2 = LeftShiftd(k7, 8);    KO3 = LeftShiftd(k8, 13);
            KI1 = k6 ^ c6;    KI2 = k5 ^ c5;    KI3 = k1 ^ c1;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 3
            KL1 = LeftShiftd(k3, 1);    KL2 = k5 ^ c5;
            KO1 = LeftShiftd(k4, 5);    KO2 = LeftShiftd(k8, 8);    KO3 = LeftShiftd(k1, 13);
            KI1 = k7 ^ c7;    KI2 = k6 ^ c6;    KI3 = k2 ^ c2;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 4
            KL1 = LeftShiftd(k4, 1);    KL2 = k6 ^ c6;
            KO1 = LeftShiftd(k5, 5);    KO2 = LeftShiftd(k1, 8);    KO3 = LeftShiftd(k2, 13);
            KI1 = k8 ^ c8;    KI2 = k7 ^ c7;    KI3 = k3 ^ c3;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 5
            KL1 = LeftShiftd(k5, 1);    KL2 = k7 ^ c7;
            KO1 = LeftShiftd(k6, 5);    KO2 = LeftShiftd(k2, 8);    KO3 = LeftShiftd(k3, 13);
            KI1 = k1 ^ c1;    KI2 = k8 ^ c8;    KI3 = k4 ^ c4;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 6
            KL1 = LeftShiftd(k6, 1);    KL2 = k8 ^ c8;
            KO1 = LeftShiftd(k7, 5);    KO2 = LeftShiftd(k3, 8);    KO3 = LeftShiftd(k4, 13);
            KI1 = k2 ^ c2;    KI2 = k1 ^ c1;    KI3 = k5 ^ c5;
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right;    in_right = in_left;    in_left = temp;
            // Round 7
            KL1 = LeftShiftd(k7, 1);    KL2 = k1 ^ c1;
            KO1 = LeftShiftd(k8, 5);    KO2 = LeftShiftd(k4, 8);    KO3 = LeftShiftd(k5, 13);
            KI1 = k3 ^ c3;    KI2 = k2 ^ c2;    KI3 = k6 ^ c6;
            temp = FLd(in_left, KL1, KL2);
            temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
            temp ^= in_right;
            
                in_right = in_left;    in_left = temp;
                // Round 8
                KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ c2;
                KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
                KI1 = k4 ^ c4;    KI2 = k3 ^ c3;    KI3 = k7 ^ c7;
                temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
                temp = FLd(temp, KL1, KL2);
                temp ^= in_right; //   in_right = in_left;    in_left = temp;
 //               if (temp == cipher_left && in_left == cipher_right) printf("The secret key is %08x%08x\n", threadIndex, i);          
                cipherl[threadIndex] = temp; cipherr[threadIndex] = in_left; // In TMTO we will be writing the results to the memory, so we have this extra step
            k8++; k4 = k8;
        }
        k7++; k3 = k7;
        /*       KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ 0x4567;
        KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
        KI1 = k4 ^ 0xCDEF;    KI2 = k3 ^ 0x89AB;    KI3 = k7 ^ 0x7654;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right; //   in_right = in_left;    in_left = temp;
        //printf("%08x%08x\n", in_left, in_right);

        if (temp == cipher_left)
            if (in_left == cipher_right)
                printf("The secret key is %08x%08x\n", threadIndex, i);
        k8++; k4 = k8;*/
    }
}
__global__ void KASUMI64ExhaustiveConstants(bit32 left, bit32 right, bit32 cipher_left, bit32 cipher_right, bit8* S7G, bit16* S9G, bit16* constantG) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 512) {
        if (threadIdx.x < 128) S7S[threadIdx.x] = S7G[threadIdx.x];
        if (threadIdx.x < 8) constants[threadIdx.x] = constantG[threadIdx.x];
        S9S[threadIdx.x] = S9G[threadIdx.x];
    }
    __syncthreads();
    bit32 in_left = left, in_right = right, temp;
    bit16 k1 = threadIndex / 65536, k2 = threadIndex % 65536, k3 = 0, k4 = 0, k5 = threadIndex / 65536, k6 = threadIndex % 65536, k7 = 0, k8 = 0;
    bit16 KL1, KL2, KO1, KO2, KO3, KI1, KI2, KI3;

#pragma unroll
    for (int j = 0; j < 16; j++) {
    for (int i = 0; i < 65536; i++) {
        in_left = left; in_right = right;
        // Round 1
        KL1 = LeftShiftd(k1, 1);    KL2 = k3 ^ constants[2];
        KO1 = LeftShiftd(k2, 5);    KO2 = LeftShiftd(k6, 8);    KO3 = LeftShiftd(k7, 13);
        KI1 = k5 ^ constants[4];    KI2 = k4 ^ constants[3];    KI3 = k8 ^ constants[7];
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 2
        KL1 = LeftShiftd(k2, 1);    KL2 = k4 ^ constants[3];
        KO1 = LeftShiftd(k3, 5);    KO2 = LeftShiftd(k7, 8);    KO3 = LeftShiftd(k8, 13);
        KI1 = k6 ^ constants[5];    KI2 = k5 ^ constants[4];    KI3 = k1 ^ constants[0];
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 3
        KL1 = LeftShiftd(k3, 1);    KL2 = k5 ^ constants[4];
        KO1 = LeftShiftd(k4, 5);    KO2 = LeftShiftd(k8, 8);    KO3 = LeftShiftd(k1, 13);
        KI1 = k7 ^ constants[6];    KI2 = k6 ^ constants[5];    KI3 = k2 ^ constants[1];
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 4
        KL1 = LeftShiftd(k4, 1);    KL2 = k6 ^ constants[5];
        KO1 = LeftShiftd(k5, 5);    KO2 = LeftShiftd(k1, 8);    KO3 = LeftShiftd(k2, 13);
        KI1 = k8 ^ constants[7];    KI2 = k7 ^ constants[6];    KI3 = k3 ^ constants[2];
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 5
        KL1 = LeftShiftd(k5, 1);    KL2 = k7 ^ constants[6];
        KO1 = LeftShiftd(k6, 5);    KO2 = LeftShiftd(k2, 8);    KO3 = LeftShiftd(k3, 13);
        KI1 = k1 ^ constants[0];    KI2 = k8 ^ constants[7];    KI3 = k4 ^ constants[3];
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 6
        KL1 = LeftShiftd(k6, 1);    KL2 = k8 ^ constants[7];
        KO1 = LeftShiftd(k7, 5);    KO2 = LeftShiftd(k3, 8);    KO3 = LeftShiftd(k4, 13);
        KI1 = k2 ^ constants[1];    KI2 = k1 ^ constants[0];    KI3 = k5 ^ constants[4];
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 7
        KL1 = LeftShiftd(k7, 1);    KL2 = k1 ^ constants[0];
        KO1 = LeftShiftd(k8, 5);    KO2 = LeftShiftd(k4, 8);    KO3 = LeftShiftd(k5, 13);
        KI1 = k3 ^ constants[2];    KI2 = k2 ^ constants[1];    KI3 = k6 ^ constants[5];
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;//    in_right = in_left;    in_left = temp;
        if (temp == cipher_right) {
            in_right = in_left;    in_left = temp;
            // Round 8
            KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ constants[1];
            KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
            KI1 = k4 ^ constants[3];    KI2 = k3 ^ constants[2];    KI3 = k7 ^ constants[6];
            temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
            temp = FLd(temp, KL1, KL2);
            temp ^= in_right; //   in_right = in_left;    in_left = temp;
            if (temp == cipher_left) printf("The secret key is %08x%08x\n", threadIndex, i);
        }



 /*       // Round 8
        KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ constants[1];
        KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
        KI1 = k4 ^ constants[3];    KI2 = k3 ^ constants[2];    KI3 = k7 ^ constants[6];
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        //printf("%08x%08x\n", in_left, in_right);

        if (in_left == cipher_left)
            if (in_right == cipher_right)
                printf("The secret key is %08x%08x\n", threadIndex, i);*/
        k8++; k4 = k8;
    }
    k7++; k3 = k7;
    }
}

/*__global__ void KASUMI64Exhaustive32Tables(bit32 left, bit32 right, bit32 cipher_left, bit32 cipher_right, bit8* S7G, bit16* S9G) {
    bit32 threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //    int warpThreadIndex = threadIdx.x & 31;
    if (threadIdx.x < 512) {
        if (threadIdx.x < 128) S7S[threadIdx.x] = S7G[threadIdx.x];
 //       for (int i=0;i<32; i++)
 //           S9S2[threadIdx.x / 4][i][threadIdx.x % 4] = S9G[threadIdx.x];
    }
    __syncthreads();

    bit32 in_left = left, in_right = right, temp;
    bit16 k1 = threadIndex / 65536, k2 = threadIndex % 65536, k3 = 0, k4 = 0, k5 = 0, k6 = 0, k7 = 0, k8 = 0;
    bit16 KL1, KL2, KO1, KO2, KO3, KI1, KI2, KI3;

    for (int i = 0; i < 65536; i++) {
        in_left = left; in_right = right;
        // Round 1
        KL1 = LeftShiftd(k1, 1);    KL2 = k3 ^ 0x89AB;
        KO1 = LeftShiftd(k2, 5);    KO2 = LeftShiftd(k6, 8);    KO3 = LeftShiftd(k7, 13);
        KI1 = k5 ^ 0xFEDC;    KI2 = k4 ^ 0xCDEF;    KI3 = k8 ^ 0x3210;
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 2
        KL1 = LeftShiftd(k2, 1);    KL2 = k4 ^ 0xCDEF;
        KO1 = LeftShiftd(k3, 5);    KO2 = LeftShiftd(k7, 8);    KO3 = LeftShiftd(k8, 13);
        KI1 = k6 ^ 0xBA98;    KI2 = k5 ^ 0xFEDC;    KI3 = k1 ^ 0x0123;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 3
        KL1 = LeftShiftd(k3, 1);    KL2 = k5 ^ 0xFEDC;
        KO1 = LeftShiftd(k4, 5);    KO2 = LeftShiftd(k8, 8);    KO3 = LeftShiftd(k1, 13);
        KI1 = k7 ^ 0x7654;    KI2 = k6 ^ 0xBA98;    KI3 = k2 ^ 0x4567;
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 4
        KL1 = LeftShiftd(k4, 1);    KL2 = k6 ^ 0xBA98;
        KO1 = LeftShiftd(k5, 5);    KO2 = LeftShiftd(k1, 8);    KO3 = LeftShiftd(k2, 13);
        KI1 = k8 ^ 0x3210;    KI2 = k7 ^ 0x7654;    KI3 = k3 ^ 0x89AB;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 5
        KL1 = LeftShiftd(k5, 1);    KL2 = k7 ^ 0x7654;
        KO1 = LeftShiftd(k6, 5);    KO2 = LeftShiftd(k2, 8);    KO3 = LeftShiftd(k3, 13);
        KI1 = k1 ^ 0x0123;    KI2 = k8 ^ 0x3210;    KI3 = k4 ^ 0xCDEF;
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 6
        KL1 = LeftShiftd(k6, 1);    KL2 = k8 ^ 0x3210;
        KO1 = LeftShiftd(k7, 5);    KO2 = LeftShiftd(k3, 8);    KO3 = LeftShiftd(k4, 13);
        KI1 = k2 ^ 0x4567;    KI2 = k1 ^ 0x0123;    KI3 = k5 ^ 0xFEDC;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 7
        KL1 = LeftShiftd(k7, 1);    KL2 = k1 ^ 0x0123;
        KO1 = LeftShiftd(k8, 5);    KO2 = LeftShiftd(k4, 8);    KO3 = LeftShiftd(k5, 13);
        KI1 = k3 ^ 0x89AB;    KI2 = k2 ^ 0x4567;    KI3 = k6 ^ 0xBA98;
        temp = FLd(in_left, KL1, KL2);
        temp = FOd(temp, KO1, KO2, KO3, KI1, KI2, KI3);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        // Round 8
        KL1 = LeftShiftd(k8, 1);    KL2 = k2 ^ 0x4567;
        KO1 = LeftShiftd(k1, 5);    KO2 = LeftShiftd(k5, 8);    KO3 = LeftShiftd(k6, 13);
        KI1 = k4 ^ 0xCDEF;    KI2 = k3 ^ 0x89AB;    KI3 = k7 ^ 0x7654;
        temp = FOd(in_left, KO1, KO2, KO3, KI1, KI2, KI3);
        temp = FLd(temp, KL1, KL2);
        temp ^= in_right;    in_right = in_left;    in_left = temp;
        //printf("%08x%08x\n", in_left, in_right);

        if (in_left == cipher_left)
            if (in_right == cipher_right)
                printf("The secret key is %08x%08x\n", threadIndex, i);
        k8++;
    }
}*/

int main(void) {
	hipSetDevice(0);
    bit32 plaintextl = 0, plaintextr=0, ciphertextl= 0xf54cfbf7, ciphertextr= 0x5f3b5699;
    // Allocate tables
    bit8 *S7d;
    bit16 *S9d;
    bit16* constantd;
    float milliseconds = 0;

    bit32* ciphertextl_d, *ciphertextr_d;
    bit32* cipherl, * cipherr;

    cipherl = (bit32*)calloc(BLOCKS * THREADS, sizeof(bit32));
    cipherr = (bit32*)calloc(BLOCKS * THREADS, sizeof(bit32));

    hipMalloc((void**)&ciphertextl_d, BLOCKS * THREADS * sizeof(bit32));
    hipMalloc((void**)&ciphertextr_d, BLOCKS * THREADS * sizeof(bit32));


    hipMallocManaged(&S7d, 128 * sizeof(bit8));
    hipMallocManaged(&constantd, 8 * sizeof(bit16));
    hipMallocManaged(&S9d, 512 * sizeof(bit16));
    for (int i = 0; i < 128; i++) S7d[i] = S7[i];
    for (int i = 0; i < 512; i++) S9d[i] = S9[i];
    for (int i = 0; i < 8; i++) constantd[i] = constant[i];

    
    hipDeviceSynchronize(); clock_t beginTime = clock();
    hipEvent_t start, stop;	hipEventCreate(&start);	hipEventCreate(&stop);	hipEventRecord(start);
//    KASUMI64Exhaustive << <BLOCKS, THREADS >> > (plaintextl, plaintextr, ciphertextl, ciphertextr, S7d, S9d);
//    KASUMI64EncryptionTMTO << <BLOCKS, THREADS >> > (plaintextl, plaintextr, ciphertextl, ciphertextr, S7d, S9d);
//    KASUMI64ExhaustiveConstants << <BLOCKS, THREADS >> > (plaintextl, plaintextr, ciphertextl, ciphertextr, S7d, S9d, constantd);
   KASUMI64ExhaustiveConstantsRegister << <BLOCKS, THREADS >> > (plaintextl, plaintextr, ciphertextl, ciphertextr, S7d, S9d); //best
//    KASUMI64ExhaustiveConstantsRegisterTMTO << <BLOCKS, THREADS >> > (plaintextl, plaintextr, ciphertextl, ciphertextr, S7d, S9d, ciphertextl_d,ciphertextr_d); //best
 //   KASUMI64Exhaustive32Tables << <BLOCKS, THREADS >> > (plaintextl, plaintextr, ciphertextl, ciphertextr, S7d, S9d);
    hipEventRecord(stop);	hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);	printf("Time elapsed: %f milliseconds ", milliseconds);

    hipMemcpy(cipherl, ciphertextl_d, BLOCKS * THREADS * sizeof(bit32), hipMemcpyDeviceToHost);
    hipMemcpy(cipherr, ciphertextr_d, BLOCKS * THREADS * sizeof(bit32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("Time elapsed: %f sec\n", float(clock() - beginTime) / CLOCKS_PER_SEC);
    printf("-------------------------------\n");
//    encryption(0,0,0xf54cfbf7,0x5f3b5699);
	printf("%s\n", hipGetErrorString(hipGetLastError()));
}

